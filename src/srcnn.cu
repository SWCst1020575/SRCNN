#include "hip/hip_runtime.h"
/*******************************************************************************
 * SRCNN: Super-Resolution with deep Convolutional Neural Networks
 * ----------------------------------------------------------------------------
 * Current Author : Raphael Kim ( rageworx@gmail.com )
 * Latest update  : 2023-03-08
 * Pre-Author     : Wang Shu
 * Origin-Date    @ Sun 13 Sep, 2015
 * Descriptin ..
 *                 This source code modified version from Origianl code of Wang
 *                Shu's. All license following from origin.
 *******************************************************************************/
#ifndef EXPORTLIBSRCNN

////////////////////////////////////////////////////////////////////////////////
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>
#ifndef NO_OMP
#include <omp.h>
#endif

#include "srcnn.h"
#include "tick.h"

/* pre-calculated convolutional data */
#include "convdata.h"
#include "convdataCuda.cuh"
////////////////////////////////////////////////////////////////////////////////

#define THREAD 256
#define BLOCK 2048

static float image_multiply = 2.0f;
static unsigned image_width = 0;
static unsigned image_height = 0;
static bool opt_verbose = true;
static bool opt_debug = false;
static bool opt_help = false;
static int t_exit_code = 0;

static std::string path_me;
static std::string file_me;
static std::string file_src;
static std::string file_dst;

////////////////////////////////////////////////////////////////////////////////

#define DEF_STR_VERSION "0.1.5.20"

////////////////////////////////////////////////////////////////////////////////

/* Function Declaration */
void Convolution99(cv::Mat& src, cv::Mat& dst,
                   const float kernel[9][9], float bias);

void Convolution11(std::vector<cv::Mat>& src, cv::Mat& dst,
                   const float kernel[CONV1_FILTERS], float bias);

__global__ void Convolution55(float* src, unsigned char* dst, int* rowf, int* colf, int height, int width);

__global__ void Convolution99x11(unsigned char* src, float* dst, int* rowf, int* colf, int height, int width);

////////////////////////////////////////////////////////////////////////////////

static inline int IntTrim(int a, int b, int c) {
    int buff[3] = {a, c, b};
    return buff[(int)(c > a) + (int)(c > b)];
}
__device__ static inline int IntTrimCuda(int a, int b, int c) {
    int buff[3] = {a, c, b};
    return buff[(int)(c > a) + (int)(c > b)];
}
/***
 * FuncName : Convolution99
 * Function : Complete one cell in the first Convolutional Layer
 * Parameter    : src - the original input image
 *        dst - the output image
 *        kernel - the convolutional kernel
 *        bias - the cell bias
 * Output   : <void>
 ***/
void Convolution99(cv::Mat& src, cv::Mat& dst, const float kernel[9][9], float bias) {
    int width = dst.cols;
    int height = dst.rows;
    int row = 0;
    int col = 0;
    // macOS clang displays these array not be initialized.
    int rowf[height + 8];
    int colf[width + 8];

/* Expand the src image */
#pragma parallel for
    for (row = 0; row < height + 8; row++) {
        rowf[row] = IntTrim(0, height - 1, row - 4);
    }

#pragma parallel for
    for (col = 0; col < width + 8; col++) {
        colf[col] = IntTrim(0, width - 1, col - 4);
    }

/* Complete the Convolution Step */
#pragma omp parallel for private(col)
    for (row = 0; row < height; row++) {
        for (col = 0; col < width; col++) {
            /* Convolution */
            float temp = 0.f;

            for (int i = 0; i < 9; i++) {
                for (int j = 0; j < 9; j++) {
                    temp += kernel[i][j] * src.at<uint8_t>(rowf[row + i], colf[col + j]);
                }
            }

            temp += bias;

            /* Threshold */
            temp = (temp < 0) ? 0 : temp;

            dst.at<float>(row, col) = temp;
        }
    }
}

/***
 * FuncName : Convolution11
 * Function : Complete one cell in the second Convolutional Layer
 * Parameter    : src - the first layer data
 *        dst - the output data
 *        kernel - the convolutional kernel
 *        bias - the cell bias
 * Output   : <void>
 ***/
void Convolution11(std::vector<cv::Mat>& src, cv::Mat& dst, const float kernel[CONV1_FILTERS], float bias) {
    int height = dst.rows;
    int width = dst.cols;
    int row = 0;
    int col = 0;

#pragma omp parallel for private(col)
    for (row = 0; row < height; row++) {
        for (col = 0; col < width; col++) {
            /* Process with each pixel */
            float temp = 0.f;

            for (int i = 0; i < CONV1_FILTERS; i++) {
                temp += src[i].at<float>(row, col) * kernel[i];
            }
            temp += bias;

            /* Threshold */
            temp = (temp < 0) ? 0 : temp;

            dst.at<float>(row, col) = temp;
        }
    }
}

/***
 * FuncName : Convolution55
 * Function : Complete the cell in the third Convolutional Layer
 * Parameter    : src - the second layer data
 *        dst - the output image
 *        kernel - the convolutional kernel
 *        bias - the cell bias
 * Output   : <void>
 ***/
__global__ void Convolution55(float* src, unsigned char* dst, int* rowf, int* colf, int height, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row = 0;
    int col = 0;

    /* Complete the Convolution Step */
    for (int id = idx; id < width * height; id += THREAD * BLOCK) {
        row = id / width;
        col = id % width;
        float temp = 0;
        for (int i = 0; i < CONV2_FILTERS; i++) {
            double temppixel = 0;
#pragma unroll
            for (int m = 0; m < 5; m++) {
#pragma unroll
                for (int n = 0; n < 5; n++) {
                    // temppixel +=
                    //     weights_conv3_data_cuda[i][m][n] * src[i].at<float>(rowf[row + m], colf[col + n]);
                    temppixel += weights_conv3_data_cuda[i][m][n] * src[i * width * height + rowf[row + m] * width + colf[col + n]];
                }
            }

            temp += temppixel;
        }

        temp += biases_conv3_cuda;

        /* Threshold */
        temp = IntTrimCuda(0, 255, temp);

        // dst.at<unsigned char>(row, col) = (unsigned char)temp;
        dst[row * width + col] = (unsigned char)temp;
    }
}

/***
 * FuncName : Convolution99x11
 * Function : Complete one cell in the first and second Convolutional Layer
 * Parameter    : src - the original input image
 *        dst - the output image
 *        kernel - the convolutional kernel
 *        bias - the cell bias
 * Output   : <void>
 ***/
__global__ void intTrimData(int* rowf, int* colf, int height, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = idx; i < height + 8; i += blockDim.x * gridDim.x)
        rowf[i] = IntTrimCuda(0, height - 1, i - 4);
    for (int i = idx; i < width + 8; i += blockDim.x * gridDim.x)
        colf[i] = IntTrimCuda(0, width - 1, i - 4);
}
__global__ void intTrimData2(int* rowf, int* colf, int height, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = idx; i < height + 4; i += blockDim.x * gridDim.x)
        rowf[i] = IntTrimCuda(0, height - 1, i - 2);
    for (int i = idx; i < width + 4; i += blockDim.x * gridDim.x)
        colf[i] = IntTrimCuda(0, width - 1, i - 2);
}
__global__ void Convolution99x11(unsigned char* src, float* dst, int* rowf, int* colf, int height, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row = 0;
    int col = 0;
    float temp[CONV1_FILTERS] = {0.f};
    /* Complete the Convolution Step */
    for (int id = idx; id < width * height; id += THREAD * BLOCK) {
        row = id / width;
        col = id % width;
        for (int k = 0; k < CONV1_FILTERS; k++) {
            /* Convolution */
            temp[k] = 0.0;
#pragma unroll
            for (int i = 0; i < 9; i++) {
#pragma unroll
                for (int j = 0; j < 9; j++) {
                    temp[k] += weights_conv1_data_cuda[k][i][j] * src[rowf[row + i] * width + colf[col + j]];
                    // temp[k] += weights_conv1_data_cuda[k][i][j] * src.at<uint8_t>(rowf[row + i], colf[col + j]);
                }
            }

            temp[k] += biases_conv1_cuda[k];

            /* Threshold */
            temp[k] = (temp[k] < 0) ? 0 : temp[k];
        }

        /* Process with each pixel */
        for (int k = 0; k < CONV2_FILTERS; k++) {
            float result = 0.0;
#pragma unroll
            for (int i = 0; i < CONV1_FILTERS; i++) {
                result += temp[i] * weights_conv2_data_cuda[k][i];
            }
            result += biases_conv2_cuda[k];

            /* Threshold */
            result = (result < 0) ? 0 : result;
            dst[k * width * height + row * width + col] = result;
            // dst[k].at<float>(row, col) = result;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

bool parseArgs(int argc, char** argv) {
    for (int cnt = 0; cnt < argc; cnt++) {
        std::string strtmp = argv[cnt];
        size_t fpos = std::string::npos;

        if (cnt == 0) {
            fpos = strtmp.find_last_of("\\");

            if (fpos == std::string::npos) {
                fpos = strtmp.find_last_of("/");
            }

            if (fpos != std::string::npos) {
                path_me = strtmp.substr(0, fpos);
                file_me = strtmp.substr(fpos + 1);
            } else {
                file_me = strtmp;
            }
        } else {
            if (strtmp.find("--scale=") == 0) {
                std::string strval = strtmp.substr(8);
                if (strval.size() > 0) {
                    float tmpfv = atof(strval.c_str());
                    if (tmpfv > 0.f) {
                        image_multiply = tmpfv;
                    }
                }
            } else if (strtmp.find("--noverbose") == 0) {
                opt_verbose = false;
            } else if (strtmp.find("--help") == 0) {
                opt_help = true;
            } else if (file_src.size() == 0) {
                file_src = strtmp;
            } else if (file_dst.size() == 0) {
                file_dst = strtmp;
            }
        }
    }

    if (!opt_help) {
        if ((file_src.size() > 0) && (file_dst.size() == 0)) {
            std::string convname = file_src;
            std::string srcext;

            // changes name without file extention.
            size_t posdot = file_src.find_last_of(".");
            if (posdot != std::string::npos) {
                convname = file_src.substr(0, posdot);
                srcext = file_src.substr(posdot);
            }

            convname += "_resized";
            if (srcext.size() > 0) {
                convname += srcext;
            }

            file_dst = convname;
        }

        if ((file_src.size() > 0) && (file_dst.size() > 0)) {
            return true;
        }
    }

    return false;
}

void printTitle() {
    printf("%s : Super-Resolution with deep Convolutional Neural Networks\n",
           file_me.c_str());
    printf("(C)2018..2023 Raphael Kim, (C)2014 Wang Shu., version %s\n",
           DEF_STR_VERSION);
    printf("Built with OpenCV version %s\n", CV_VERSION);
}

void printHelp() {
    printf("\n");
    printf("    usage : %s (options) [source file name] ([output file name])\n", file_me.c_str());
    printf("\n");
    printf("    _options_:\n");
    printf("\n");
    printf("        --scale=( ratio: 0.1 to .. ) : scaling by ratio.\n");
    printf("        --noverbose                  : turns off all verbose\n");
    printf("        --help                       : this help\n");
    printf("\n");
}

void* pthreadcall(void* p) {
    if (opt_verbose == true) {
        printTitle();
        printf("\n");
        printf("- Scale multiply ratio : %.2f\n", image_multiply);
        fflush(stdout);
    }

    /* Read the original image */
    cv::Mat pImgOrigin;
    pImgOrigin = cv::imread(file_src.c_str());

    if (pImgOrigin.empty() == false) {
        if (opt_verbose == true) {
            printf("- Image load : %s\n", file_src.c_str());
            fflush(stdout);
        }
    } else {
        if (opt_verbose == true) {
            printf("- load failure : %s\n", file_src.c_str());
        }

        t_exit_code = -1;
        pthread_exit(&t_exit_code);
    }

    // Test image resize target ...
    cv::Size testsz = pImgOrigin.size();
    if ((((float)testsz.width * image_multiply) <= 0.f) ||
        (((float)testsz.height * image_multiply) <= 0.f)) {
        if (opt_verbose == true) {
            printf("- Image scale error : ratio too small.\n");
        }

        t_exit_code = -1;
        pthread_exit(&t_exit_code);
    }

    // -------------------------------------------------------------

    if (opt_verbose == true) {
        printf("- Image converting to Y-Cr-Cb : ");
        fflush(stdout);
    }

    unsigned perf_tick0 = tick::getTickCount();

    /* Convert the image from BGR to YCrCb Space */
    cv::Mat pImgYCrCb;
    auto start = tick::getCurrent();
    cvtColor(pImgOrigin, pImgYCrCb, CV_BGR2YCrCb);
    auto end = tick::getCurrent();
    if (pImgYCrCb.empty() == false) {
        if (opt_verbose == true) {
            printf("Ok. %u us\n", tick::getDiff(start, end));
            fflush(stdout);
        }
    } else {
        if (opt_verbose == true) {
            printf("Failure.\n");
        }

        t_exit_code = -2;
        pthread_exit(&t_exit_code);
    }

    // ------------------------------------------------------------

    if (opt_verbose == true) {
        printf("- Splitting channels : ");
        fflush(stdout);
    }

    /* Split the Y-Cr-Cb channel */
    std::vector<cv::Mat> pImgYCrCbCh(3);
    start = tick::getCurrent();
    split(pImgYCrCb, pImgYCrCbCh);
    end = tick::getCurrent();
    if (pImgYCrCb.empty() == false) {
        if (opt_verbose == true) {
            printf("Ok. %u us\n", tick::getDiff(start, end));
            fflush(stdout);
        }
    } else {
        if (opt_verbose == true) {
            printf("Failure.\n");
            t_exit_code = -3;
            pthread_exit(&t_exit_code);
        }
    }

    // ------------------------------------------------------------

    if (opt_verbose == true) {
        printf("- Resizing splitted channels with bicublic interpolation : ");
    }

    /* Resize the Y-Cr-Cb Channel with Bicubic Interpolation */
    std::vector<cv::Mat> pImg(3);
    start = tick::getCurrent();
#pragma omp parallel for
    for (int i = 0; i < 3; i++) {
        cv::Size newsz = pImgYCrCbCh[i].size();
        newsz.width *= image_multiply;
        newsz.height *= image_multiply;

        resize(pImgYCrCbCh[i],
               pImg[i],
               newsz,
               0,
               0,
               CV_INTER_CUBIC);
    }
    end = tick::getCurrent();
    if (opt_verbose == true) {
        printf("Ok. %u us\n", tick::getDiff(start, end));
    }

    // -----------------------------------------------------------

    int cnt = 0;

    /******************* The First Layer *******************/

    if (opt_verbose == true) {
        printf("- Processing convolutional layer I + II ... ");
        fflush(stdout);
    }

    std::vector<cv::Mat> pImgConv2(CONV2_FILTERS);
    start = tick::getCurrent();

    // #pragma omp parallel for
    //     for (unsigned cnt = 0; cnt < CONV2_FILTERS; cnt++) {
    //         pImgConv2[cnt].create(pImg[0].size(), CV_32F);
    //     }
    end = tick::getCurrent();
    printf("\n  create: %u us.", tick::getDiff(start, end));

    // first conv

    unsigned char* srcImg;
    float* firstConv;
    int* rowf;
    int* colf;
    unsigned char* dstImg;

    hipMalloc(&rowf, (pImg[0].cols + 8) * sizeof(int));
    hipMalloc(&colf, (pImg[0].rows + 8) * sizeof(int));
    
    printf("\ncuda malloc complete\n");

    hipMemcpy(srcImg, pImg[0].data, pImg[0].cols * pImg[0].rows * sizeof(unsigned char), hipMemcpyHostToDevice);
    printf("cuda memcpy complete\n");

    intTrimData<<<16, THREAD>>>(rowf, colf, pImg[0].rows, pImg[0].cols);

    hipMalloc(&srcImg, pImg[0].cols * pImg[0].rows * sizeof(unsigned char));
    hipMalloc(&firstConv, pImg[0].cols * pImg[0].rows * sizeof(float) * CONV2_FILTERS);
    printf("cuda malloc complete\n");
    hipDeviceSynchronize();
    printf("intTrim init complete\n");

    start = tick::getCurrent();
    Convolution99x11<<<BLOCK, THREAD>>>(srcImg, firstConv, rowf, colf, pImg[0].rows, pImg[0].cols);
    hipMalloc(&dstImg, pImg[0].cols * pImg[0].rows * sizeof(unsigned char));
    printf("cuda malloc complete\n");
    hipDeviceSynchronize();
    end = tick::getCurrent();
    printf("cuda Convolution99x11 complete\n");

    printf("\n  Convolution: %u us.\n", tick::getDiff(start, end));
    if (opt_verbose == true) {
        printf("completed.\n");
        fflush(stdout);
    }

    /******************* The Third Layer *******************/

    if (opt_verbose == true) {
        printf("- Processing convolutional layer III ... ");
        fflush(stdout);
    }
    // second conv

    cv::Mat pImgConv3;
    pImgConv3.create(pImg[0].size(), CV_8U);
    start = tick::getCurrent();
    intTrimData2<<<16, THREAD>>>(rowf, colf, pImg[0].rows, pImg[0].cols);

    hipDeviceSynchronize();
    printf("\nintTrim init complete\n");

    

    start = tick::getCurrent();
    Convolution55<<<BLOCK, THREAD>>>(firstConv, dstImg, rowf, colf, pImg[0].rows, pImg[0].cols);
    hipDeviceSynchronize();
    printf("cuda Convolution55 complete\n");
    end = tick::getCurrent();

    unsigned char* convImg = (unsigned char*)malloc(pImg[0].size().area() * sizeof(unsigned char));
    hipMemcpy(convImg, dstImg, pImg[0].cols * pImg[0].rows * sizeof(unsigned char), hipMemcpyDeviceToHost);
    pImgConv3 = cv::Mat(pImg[0].size(), CV_8U, convImg).clone();
    free(convImg);

    printf("\n  Convolution: %u us.\n", tick::getDiff(start, end));
    if (opt_verbose == true) {
        printf("completed.\n");
        printf("- Merging images : ");
        fflush(stdout);
    }
    hipDeviceReset();
    /* Merge the Y-Cr-Cb Channel into an image */
    start = tick::getCurrent();
    cv::Mat pImgYCrCbOut;
    pImg[0] = pImgConv3;
    merge(pImg, pImgYCrCbOut);
    end = tick::getCurrent();
    if (opt_verbose == true) {
        printf("Ok. %u us.\n", tick::getDiff(start, end));
        fflush(stdout);
    }

    // ---------------------------------------------------------

    if (opt_verbose == true) {
        printf("- Converting channel to BGR : ");
        fflush(stdout);
    }

    /* Convert the image from YCrCb to BGR Space */
    cv::Mat pImgBGROut;
    start = tick::getCurrent();
    cvtColor(pImgYCrCbOut, pImgBGROut, CV_YCrCb2BGR);
    end = tick::getCurrent();
    unsigned perf_tick1 = tick::getTickCount();

    if (pImgBGROut.empty() == false) {
        if (opt_verbose == true) {
            printf("Ok. %u us.\n", tick::getDiff(start, end));
            printf("- Writing result to %s : ", file_dst.c_str());
            fflush(stdout);
        }

        imwrite(file_dst.c_str(), pImgBGROut);

        if (opt_verbose == true) {
            printf("Ok.\n");
        }
    } else {
        if (opt_verbose == true) {
            printf("Failure.\n");
        }

        t_exit_code = -10;
        pthread_exit(&t_exit_code);
    }

    if (opt_verbose == true) {
        printf("- Performace : %u ms took.\n", perf_tick1 - perf_tick0);
    }

    fflush(stdout);

    t_exit_code = 0;
    pthread_exit(NULL);
    return NULL;
}

/***
 * FuncName : main
 * Function : the entry of the program
 * Parameter    : argc - the number of the initial parameters
 *        argv - the entity of the initial parameters
 * Output   : int 0 for normal / int 1 for failed
 ***/
int main(int argc, char** argv) {
    if (parseArgs(argc, argv) == false) {
        printTitle();
        printHelp();
        fflush(stdout);
        return 0;
    }

    pthread_t ptt;
    int tid = 0;
    if (pthread_create(&ptt, NULL, pthreadcall, &tid) == 0) {
        // Wait for thread ends ..
        pthread_join(ptt, NULL);
    } else {
        printf("Error: pthread failure.\n");
    }

    return t_exit_code;
}
#endif  /// of EXPORTLIBSRCNN
